/*
Author: Stella Fournier
Class: ECE4122 A
Last Date Modified: 11/1/2022

Description:

This is the CUDA program for the thin plates calculation over number of iterations.
It takes in the whole grid size and the number of iterations from the command line and outputs
a csv text file of the new temperatures of the whole grid.

*/

#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <cmath>
#include <fstream>
#include <string>
#include <stdio.h>
#include <chrono>

using namespace std;

// Iterate thin plates temperature in GPU using CUDA
__global__ void iterateTemp(double *myPlate, double *myNewPlate, int n)
{
    int num = blockIdx.x * blockDim.x + threadIdx.x;
    int row = num % n;
    int col = num / n;
    if (num < (n * n) && (col > 0 && col < n - 1) && (row > 0 && row < n - 1))
    {
        myNewPlate[n * row + col] = 0.25 * (myPlate[n * (row - 1) + col] + myPlate[n * (row + 1) + col] + myPlate[n * row + (col - 1)] + myPlate[n * row + (col + 1)]);
    }
}

// Global Functions
// Initializing the matrix plate temperatures
void initializePlateTemp(double myPlate[], double myNewPlate[], int n)
{
    for (int row = 0; row < n; row++)
    {
        for (int col = 0; col < n; col++)
        {
            myPlate[n * row + col] = 0.0;
            myNewPlate[n * row + col] = 0.0;
            // Edges for 20 Celcius degree
            if (row == 0 || row == n - 1 || col == 0 || col == n - 1)
            {
                myPlate[n * row + col] = 20.0;
                myNewPlate[n * row + col] = 20.0;
            }
            // The 4ft 100 Celcius degree edge on the top thin plate
            if (col > 0.3 * (n - 1) && col < 0.7 * (n - 1) && row == 0)
            {
                myPlate[n * row + col] = 100.0;
                myNewPlate[n * row + col] = 100.0;
            }
        }
    }
}
// Outfile to a csv file
void outFile(double myNewPlate[], int n)
{
    fstream outputFile;
    outputFile.open("finalTemperatures.csv", ios::out | ios::app);

    outputFile << fixed << setprecision(6);

    for (int row = 0; row < n; row++)
    {
        for (int col = 0; col < n; col++)
        {
            outputFile << fixed << setw(n);
            if (col < n - 1)
            {
                outputFile << myNewPlate[n * row + col] << ",";
            }
            else
            {
                outputFile << myNewPlate[n * row + col];
            }
        }
        outputFile << endl;
    }
    outputFile.close();
}
// Checks if input is valid or not
bool validOrNot(string myString)
{
    if (myString.empty())
    {
        return false;
    }
    if (myString.length() > 1 && myString[0] == '0')
    {
        return false;
    }
    // checks if it is a digit or if it has alphabets
    for (int i = 0; i < myString.length(); i++)
    {
        if (!isdigit(myString[i]) || isalpha(myString[i]))
        {
            return false;
        }
        else if (' ' == myString[i])
        {
            return false;
        }
    }
    return true;
}

int main(int argc, const char *argv[])
{
    // checks if input is the correct format or not
    if (argc < 4 || argc > 5)
    {
        cout << "Invalid parameters, please check your values." << endl;
        return 1;
    }
    // checks if input is valid or not
    if (validOrNot(argv[2]) && validOrNot(argv[4]))
    {
        int dimensions;
        int innerDimensions;
        long iterations;
        innerDimensions = sqrt(stoi(argv[2], nullptr, 10));
        iterations = stoi(argv[4], nullptr, 10);
        dimensions = innerDimensions + 2;
        // Checks if number of dimensions and iterations is good or not
        if (dimensions < 0 || iterations < 0)
        {
            cout << "Invalid parameters, please check your values." << endl;
            return 1;
        }

        // Initialize new and old plates
        int size = (dimensions * dimensions) * sizeof(double);
        double *currentPlate;
        double *newPlate;
        hipMallocManaged(&currentPlate, size);
        hipMallocManaged(&newPlate, size);
        initializePlateTemp(currentPlate, newPlate, dimensions);

        // iterate using GPU loop
        int dev = 0;
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);
        int numThreads = deviceProp.maxThreadsPerBlock;
        int blockSize = (((dimensions * dimensions) + numThreads - 1) / numThreads);

        // timer
        float time;
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);

        for (int i = 0; i < iterations; i++)
        {
            iterateTemp<<<blockSize, numThreads>>>(currentPlate, newPlate, dimensions);
            hipDeviceSynchronize(); // wait for GPU threads to finish
            hipMemcpy(currentPlate, newPlate, size, hipMemcpyDeviceToDevice);
        }

        // finishing timer
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);
        cout << "\nThin plate calculation took " << fixed << setprecision(2) << time << " milliseconds." << endl;

        // output file
        outFile(newPlate, dimensions);

        // free gpu
        hipFree(currentPlate);
        hipFree(newPlate);
    }
    else
    {
        cout << "Invalid parameters, please check your values." << endl;
        return 1;
    }

    return 0;
}